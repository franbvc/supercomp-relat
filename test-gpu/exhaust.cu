#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/transform_reduce.h>
#include <thrust/copy.h>
#include <iostream>
#include <math.h>
#include "cuda_utils.h"

using namespace std;

struct validate
{
    const int movieCount;
    thrust::device_ptr<const int> movieStart;
    thrust::device_ptr<const int> movieEnd;
    thrust::device_ptr<const int> movieCat;

    validate(
        const int movieCount_,
        thrust::device_ptr<const int> movieStart_,
        thrust::device_ptr<const int> movieEnd_,
        thrust::device_ptr<const int> movieCat_)
        : movieCount(movieCount_),
          movieStart(movieStart_),
          movieEnd(movieEnd_),
          movieCat(movieCat_){};

    __host__ __device__ int operator()(const int &bin)
    {
        int lastIndex = -1;
        int result = 0;

        for (int i = 0; i < movieCount; i++)
        {
            bool bit = (bin & (1 << i)) != 0;
            if (!bit)
                continue;

            if (lastIndex == -1)
            {
                lastIndex = i;
                result++;
                continue;
            }

            if (movieEnd[lastIndex] > movieStart[i])
                return 0;

            lastIndex = i;
            result++;
        }

        return result;
    }
};

int main()
{
    vector<int> catLimit;
    vector<int> movieStart;
    vector<int> movieEnd;
    vector<int> movieCat;

    int movieCount = fillDataVectors(catLimit, movieStart, movieEnd, movieCat);
    thrust::device_vector<int> devCatLimit(catLimit);
    thrust::device_vector<int> devStart(movieStart);
    thrust::device_vector<int> devEnd(movieEnd);
    thrust::device_vector<int> devCat(movieCat);

    thrust::counting_iterator<int> startBin(1);
    thrust::counting_iterator<int> endBin = startBin + (int)pow(2, movieCount) - 1;

    validate validateFunctor(movieCount, devStart.data(), devEnd.data(), devCat.data());

    thrust::device_vector<int> validCombinations;
    thrust::copy_if(startBin, endBin, validCombinations.begin(), validateFunctor);

    for (auto el : validCombinations)
    {
        cout << el << " ";
    }

    return 0;
}
