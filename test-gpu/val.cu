#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/transform_reduce.h>
#include <iostream>
#include <math.h>
#include "cuda_utils.h"

using namespace std;

struct validate
{
    const int movieCount;
    // thrust::device_vector<int> catLimit;
    thrust::device_ptr<const int> catLimit;
    thrust::device_ptr<const int> movieStart;
    thrust::device_ptr<const int> movieEnd;
    thrust::device_ptr<const int> movieCat;

    validate(
        const int movieCount_,
        // const thrust::device_vector<int> &catLimit_,
        thrust::device_ptr<const int> catLimit_,
        thrust::device_ptr<const int> movieStart_,
        thrust::device_ptr<const int> movieEnd_,
        thrust::device_ptr<const int> movieCat_)
        : movieCount(movieCount_),
          catLimit(catLimit_),
          movieStart(movieStart_),
          movieEnd(movieEnd_),
          movieCat(movieCat_){};

    __host__ __device__ int operator()(const int &binPrevious, const int &bin)
    {
        int lastIndex = -1;
        int result = 0;
        int localCatArr[20] = {0};

        for (int i = 0; i < movieCount; i++)
        {
            bool bit = (bin & (1 << i)) != 0;
            if (!bit)
                continue;

            if (lastIndex == -1)
            {
                lastIndex = i;
                result++;
                continue;
            }

            // if (localCatArr[movieCat[i] - 1] == catLimit[movieCat[i] - 1])
            // {
            //     result = 0;
            //     break;
            // }

            if (movieEnd[lastIndex] > movieStart[i])
            {
                result = 0;
                break;
            }

            // localCatArr[movieCat[i] - 1]++;
            lastIndex = i;
            result++;
        }

        return (result > binPrevious) ? result : binPrevious;
    }
};

int main()
{
    vector<int> catLimit;
    vector<int> movieStart;
    vector<int> movieEnd;
    vector<int> movieCat;

    int movieCount = fillDataVectors(catLimit, movieStart, movieEnd, movieCat);
    thrust::device_vector<int> devCatLimit(catLimit);
    thrust::device_vector<int> devStart(movieStart);
    thrust::device_vector<int> devEnd(movieEnd);
    thrust::device_vector<int> devCat(movieCat);

    thrust::counting_iterator<int> startBin(1);
    thrust::counting_iterator<int> endBin = startBin + (int)pow(2, movieCount) - 1;

    // validate validateFunctor(movieCount, devCatLimit.data(), devStart.data(), devEnd.data(), devCat.data());
    // validate validateFunctor(movieCount, devStart.data(), devEnd.data(), devCat.data());

    for (auto it = startBin; it != endBin; it++)
    {
        int bin = *it;
        int lastIndex = -1;
        int result = 0;
        int localCatArr[20] = {0};

        for (int i = 0; i < movieCount; i++)
        {
            bool bit = (bin & (1 << i)) != 0;
            if (!bit)
                continue;

            if (lastIndex == -1)
            {
                lastIndex = i;
                result++;
                continue;
            }

            // if (localCatArr[movieCat[i] - 1] == catLimit[movieCat[i] - 1])
            // {
            //     result = 0;
            //     break;
            // }

            if (movieEnd[lastIndex] > movieStart[i])
            {
                result = 0;
                break;
            }

            // localCatArr[movieCat[i] - 1]++;
            lastIndex = i;
            result++;
        }

        cout << bin << " " << result << endl;
    }

    int maxMovies = thrust::reduce(startBin, endBin, 0, validateFunctor);

    cout << maxMovies;

    return 0;
}
